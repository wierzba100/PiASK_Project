#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <cmath>
#include <cstdio>
#include <png.h>

#define TILE_SIZE 16

__constant__ int Gx_cuda[3][3] = {
    {-1, 0, 1},
    {-2, 0, 2},
    {-1, 0, 1}
};

__constant__ int Gy_cuda[3][3] = {
    {-1, -2, -1},
    { 0,  0,  0},
    { 1,  2,  1}
};

unsigned char *convert_to_grayscale(const char *filename, unsigned int *out_width, unsigned int *out_height) {
    FILE *fp = fopen(filename, "rb");
    if (!fp)
    {
        perror("Error opening file");
        exit(EXIT_FAILURE);
    }

    png_structp png = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png)
    {
        fclose(fp);
        fprintf(stderr, "Failed to create PNG read struct\n");
        exit(EXIT_FAILURE);
    }

    png_infop info = png_create_info_struct(png);
    if (!info)
    {
        png_destroy_read_struct(&png, NULL, NULL);
        fclose(fp);
        fprintf(stderr, "Failed to create PNG info struct\n");
        exit(EXIT_FAILURE);
    }

    if (setjmp(png_jmpbuf(png)))
    {
        png_destroy_read_struct(&png, &info, NULL);
        fclose(fp);
        fprintf(stderr, "Error during PNG reading\n");
        exit(EXIT_FAILURE);
    }

    png_init_io(png, fp);
    png_read_info(png, info);

    int width = png_get_image_width(png, info);
    int height = png_get_image_height(png, info);
    png_byte color_type = png_get_color_type(png, info);
    png_byte bit_depth = png_get_bit_depth(png, info);

    if (color_type == PNG_COLOR_TYPE_PALETTE)
        png_set_palette_to_rgb(png);

    if (color_type == PNG_COLOR_TYPE_GRAY && bit_depth < 8)
        png_set_expand_gray_1_2_4_to_8(png);

    if (bit_depth == 16)
        png_set_strip_16(png);

    if (color_type == PNG_COLOR_TYPE_RGBA || color_type == PNG_COLOR_TYPE_RGB_ALPHA)
        png_set_strip_alpha(png);

    png_read_update_info(png, info);

    png_bytep *row_pointers = (png_bytep *)malloc(sizeof(png_bytep) * height);
    for (int y = 0; y < height; y++)
    {
        row_pointers[y] = (png_bytep)malloc(png_get_rowbytes(png, info));
    }

    png_read_image(png, row_pointers);

    unsigned char **gray_image = (png_bytep *)malloc(height * sizeof(unsigned char *));
    for (int y = 0; y < height; y++)
    {
        gray_image[y] = (png_bytep)malloc(width * sizeof(unsigned char));
        for (int x = 0; x < width; x++) {
            png_bytep px = &(row_pointers[y][x * 3]);
            gray_image[y][x] = (unsigned char)(0.299 * px[0] + 0.587 * px[1] + 0.114 * px[2]);
        }
    }

    for (int y = 0; y < height; y++)
    {
        free(row_pointers[y]);
    }
    free(row_pointers);

    png_destroy_read_struct(&png, &info, NULL);
    fclose(fp);

    *out_width = width;
    *out_height = height;

    unsigned char* gray_image_flat = (unsigned char*)malloc(width * height * sizeof(unsigned char));
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            gray_image_flat[i * width + j] = gray_image[i][j];
        }
    }

    for (int y = 0; y < height; y++) {
        free(gray_image[y]);
    }
    free(gray_image);

    return gray_image_flat;
}

void convert_to_png(const char *filename, unsigned char *gray_image, unsigned int width, unsigned int height) {
    FILE *fp = fopen(filename, "wb");
    if (!fp) {
        perror("Error opening file for writing");
        exit(EXIT_FAILURE);
    }

    png_structp png = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png) {
        fclose(fp);
        fprintf(stderr, "Failed to create PNG write struct\n");
        exit(EXIT_FAILURE);
    }

    png_infop info = png_create_info_struct(png);
    if (!info) {
        png_destroy_write_struct(&png, NULL);
        fclose(fp);
        fprintf(stderr, "Failed to create PNG info struct\n");
        exit(EXIT_FAILURE);
    }

    if (setjmp(png_jmpbuf(png))) {
        png_destroy_write_struct(&png, &info);
        fclose(fp);
        fprintf(stderr, "Error during PNG writing\n");
        exit(EXIT_FAILURE);
    }

    png_init_io(png, fp);

    png_set_IHDR(
        png, info, width, height,
        8, PNG_COLOR_TYPE_GRAY, PNG_INTERLACE_NONE,
        PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT
    );

    png_write_info(png, info);

    png_bytep *row_pointers = (png_bytep *)malloc(sizeof(png_bytep) * height);
    for (int y = 0; y < height; y++) {
        row_pointers[y] = (png_bytep)malloc(width * sizeof(unsigned char));
        for (int x = 0; x < width; x++) {
            row_pointers[y][x] = gray_image[y * width + x]; // 1D array access
        }
    }

    png_write_image(png, row_pointers);

    for (int y = 0; y < height; y++) {
        free(row_pointers[y]);
    }
    free(row_pointers);

    png_write_end(png, NULL);

    png_destroy_write_struct(&png, &info);
    fclose(fp);
}

__global__ void sobel_filter_kernel(unsigned char *gray_image, unsigned char *output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1)
    {
        int gradient_x = 0;
        int gradient_y = 0;

        for (int i = -1; i <= 1; i++)
        {
            for (int j = -1; j <= 1; j++)
            {
                int pixel = gray_image[(y + i) * width + (x + j)];
                gradient_x += pixel * Gx_cuda[i + 1][j + 1];
                gradient_y += pixel * Gy_cuda[i + 1][j + 1];
            }
        }

        int magnitude = (int)sqrtf(gradient_x * gradient_x + gradient_y * gradient_y);
        magnitude = (magnitude > 255) ? 255 : (magnitude < 0 ? 0 : magnitude);

        output[y * width + x] = (unsigned char)magnitude;
    }
}

int main(int argc, char *argv[])
{
    if (argc != 2)
    {
        fprintf(stderr, "Usage: %s <file_image.png>\n", argv[0]);
        return EXIT_FAILURE;
    }

    unsigned int width, height;
    unsigned char *gray_image = convert_to_grayscale(argv[1], &width, &height);

    unsigned char* d_gray_image;
    unsigned char* d_output;

    hipMalloc((void**)&d_gray_image, width * height * sizeof(unsigned char));
    hipMalloc((void**)&d_output, width * height * sizeof(unsigned char));

    hipMemcpy(d_gray_image, gray_image, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 blockDim(TILE_SIZE, TILE_SIZE, 1);
    dim3 gridDim((width + TILE_SIZE - 1) / TILE_SIZE, (height + TILE_SIZE - 1) / TILE_SIZE, 1);

    sobel_filter_kernel<<<gridDim, blockDim>>>(d_gray_image, d_output, width, height);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    unsigned char* output = (unsigned char*)malloc(width * height * sizeof(unsigned char));
    hipMemcpy(output, d_output, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    convert_to_png("../Images/output_CUDA.png", output, width, height);

    hipFree(d_gray_image);
    hipFree(d_output);
    free(gray_image);
    free(output);

    return EXIT_SUCCESS;
}
